#include "hip/hip_runtime.h"
#include "particles.h"

const float gravConst = 0.0125f;

///
/// Returns radius defined by mass and density
///
float getRadius(__global Particle *p)
{
	float area = p->mass / p->density;
	return sqrt(area / PI);
}

///
/// Particle simulation step 
///
void __kernel simulate(
	__global Particle *particles,
	int particleCount,
	float stepWidth)
{
	int i;
	int id = get_global_id(0);
	if(id >= particleCount)
		return;
	if(particles[id].type == 0)
		return;
	
	__global Particle *p = &particles[id];

	float radius = getRadius(p);

	
	float2 force = (float2)(0, 0);
	for(i = 0; i < particleCount; i++)
	{
		// Ignore ourselfs
		if(i == id)
			continue;

		// Ignore empty particles
		if(particles[i].type == 0)
			continue;
			
		float2 dir = particles[i].position - p->position;
		float dist = length(dir);

		if((particles[i].flags & PARTICLE_CRUNCHER) != 0)
		{
			// Check if we ran into a particle cruncher (simple circle/point intersection)
			if(dist < (max(getRadius(&particles[i]), radius)))
			{
				bool getCrunch = true;
				// Check if we are a particle cruncher and heavier...
				if((p->flags & PARTICLE_CRUNCHER) != 0 && p->mass >= particles[i].mass)
				{
					// Prevent getting chruched
					if(p->mass == particles[i].mass)
					{
						// We have same mass, let the first survive
						if(i > id)
							getCrunch = false;
					}
					else
					{
						// We are heavier, we survive
						getCrunch = false;
					}
				}

				if(getCrunch)
				{
					p->type = 0;

					// Add velocity delta with conservation of momentum
					particles[i].velocity += (particles[i].velocity * particles[i].mass + p->velocity * p->mass) / (particles[i].mass + p->mass) - particles[i].velocity;
					particles[i].mass += p->mass;
					return;
				}
			}
		}

		// Ignore all nonattracting particles from here (gravity calculation)
		if((particles[i].flags & PARTICLE_ATTRACTOR) == 0)
			continue;

		if(dist == 0)
			continue;

		dir = normalize(dir);

		// Calculate gravitational force and add it
		force += gravConst * ((particles[i].mass * p->mass) / dist) * dir;
	}

	// Check if the particle allows movement
	if((p->flags & PARTICLE_FIXED) == 0)
	{
		// Accelerate the particle
		p->velocity += (force / p->mass) * (1 / 60.0f) * stepWidth;

		// Then move it
		p->position += p->velocity;
	}
}

///
/// Renders all particles to the backbuffer
///
void __kernel render_particles(
	__global float4 *image,
	int width,
	int height,
	int pitch,
	__global Particle *particles,
	int particleCount,
	float zoom)
{
	int id = get_global_id(0);
	if(id >= particleCount)
		return;
	if(particles[id].type == 0)
		return;
	int px = (int)(0.5f * width + zoom * particles[id].position.x);
	int py = (int)(0.5f * height + zoom * particles[id].position.y);
	int radius = max((int)(zoom * getRadius(&particles[id]) + 0.5), 0);
	
	for(int x = px - radius; x <= px + radius; x++)
	{
		for(int y = py - radius; y <= py + radius; y++)
		{
			if(x < 0 || y < 0)
				continue;
			if(x >= width || y >= height)
				continue;
			if(sqrt((float)(x-px)*(x-px) + (y-py)*(y-py)) > radius)
				continue;
			if(particles[id].flags & PARTICLE_FIXED)
				image[pitch * y + x] = (float4)(1.0f, 0.0f, 0.0f, 1.0f);
			else
				image[pitch * y + x] = (float4)(1.0f, 1.0f, 1.0f, 1.0f);
		}
	}
}

///
/// Clears the back buffer to black
///
void __kernel render_background(
	__global float4 *image,
	int width,
	int height,
	int pitch)
{
	int x = get_global_id(0);
	int y = get_global_id(1);
	if(x >= width || y >= height)
		return;
	float4 color = (float4)(0.0f, 0.0f, 0.0f, 1.0f);
	image[pitch * y + x] = color;
}